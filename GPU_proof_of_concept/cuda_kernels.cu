#include "hip/hip_runtime.h"
#include "cuda_kernels.cuh"
#include <stdio.h>

__global__ void zcx_kernel(uint64_t *x_table, uint64_t *z_table, 
                          const size_t *controls, const size_t *targets,
                          size_t num_ops, size_t num_words) {
    size_t op_idx = blockIdx.x;
    
    // Grid stride loop over words
    for (size_t word_idx = threadIdx.x; word_idx < num_words; word_idx += blockDim.x) {
        if (op_idx < num_ops) {
            size_t control = controls[op_idx];
            size_t target = targets[op_idx];
            
            z_table[control * num_words + word_idx] ^= z_table[target * num_words + word_idx];
            x_table[target * num_words + word_idx] ^= x_table[control * num_words + word_idx];
        }
    }
}

__global__ void hadamard_kernel(uint64_t *x_table, uint64_t *z_table,
                               const size_t *targets, size_t num_ops, 
                               size_t num_words) {
    size_t op_idx = blockIdx.x;
    
    // Grid stride loop over words
    for (size_t word_idx = threadIdx.x; word_idx < num_words; word_idx += blockDim.x) {
        if (op_idx < num_ops) {
            size_t target = targets[op_idx];
            size_t idx = target * num_words + word_idx;
            
            uint64_t temp = x_table[idx];
            x_table[idx] = z_table[idx];
            z_table[idx] = temp;
        }
    }
}

__global__ void measure_kernel(uint64_t *x_table, uint64_t *measurements,
                             const size_t *targets, size_t num_ops,
                             size_t num_words) {
    size_t op_idx = blockIdx.x;
    
    // Grid stride loop over words
    for (size_t word_idx = threadIdx.x; word_idx < num_words; word_idx += blockDim.x) {
        if (op_idx < num_ops) {
            size_t target = targets[op_idx];
            size_t idx = target * num_words + word_idx;
            
            measurements[idx] = x_table[idx];
            x_table[idx] = 0;
        }
    }
}

__global__ void batch_operations_kernel(uint64_t *x_table, uint64_t *z_table, uint64_t *measurements,
                                      const size_t *controls, const size_t *targets,
                                      const OpType *op_types, size_t num_ops, size_t num_words,
                                      bool debug_output) {
    size_t op_idx = blockIdx.x;
    
    // Grid stride loop over words
    for (size_t word_idx = threadIdx.x; word_idx < num_words; word_idx += blockDim.x) {
        if (op_idx < num_ops) {
            OpType op_type = op_types[op_idx];
            size_t target = targets[op_idx];
            
            if (debug_output && threadIdx.x == 0) {
                printf("Processing operation %lu: type=%d target=%lu\n", 
                       op_idx, (int)op_type, target);
            }
            
            switch(op_type) {
                case OpType::ZCX: {
                    size_t control = controls[op_idx];
                    z_table[control * num_words + word_idx] ^= z_table[target * num_words + word_idx];
                    x_table[target * num_words + word_idx] ^= x_table[control * num_words + word_idx];
                    break;
                }
                case OpType::H: {
                    size_t idx = target * num_words + word_idx;
                    uint64_t temp = x_table[idx];
                    x_table[idx] = z_table[idx];
                    z_table[idx] = temp;
                    break;
                }
                case OpType::M: {
                    size_t idx = target * num_words + word_idx;
                    measurements[idx] = x_table[idx];
                    x_table[idx] = 0;
                    break;
                }
            }
        }
    }
} 